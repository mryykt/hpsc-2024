
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>

__global__ void bucket_sort(int *bucket, int *key, int range)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(&bucket[key[i]], 1);
  __syncthreads();
  for (int i = 0, j = 0; i < range; i++)
  {
    for (; bucket[i] > 0; bucket[i]--)
    {
      key[j++] = i;
    }
  }
}

int main()
{
  int n = 50;
  int range = 5;
  // std::vector<int> key(n);
  int *key, *bucket;
  hipMallocManaged(&bucket, range * sizeof(int));
  hipMallocManaged(&key, n * sizeof(int));
  for (int i = 0; i < n; i++)
  {
    key[i] = rand() % range;
    printf("%d ", key[i]);
  }
  printf("\n");

  /*std::vector<int> bucket(range);
  for (int i=0; i<range; i++) {
    bucket[i] = 0;
  }
  for (int i=0; i<n; i++) {
    bucket[key[i]]++;
  }
  for (int i=0, j=0; i<range; i++) {
    for (; bucket[i]>0; bucket[i]--) {
      key[j++] = i;
    }
  }*/
  for (int i = 0; i < range; i++)
  {
    bucket[i] = 0;
  }
  bucket_sort<<<1, n>>>(bucket, key, range);
  hipDeviceSynchronize();

  for (int i = 0; i < n; i++)
  {
    printf("%d ", key[i]);
  }
  printf("\n");
  hipFree(bucket);
  hipFree(key);
}
