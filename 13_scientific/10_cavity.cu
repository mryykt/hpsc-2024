
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NX 41
#define NY 41
#define NT 500
#define NIT 50
#define DX (2.0 / (NX - 1))
#define DY (2.0 / (NY - 1))
#define DT 0.01
#define RHO 1.0
#define NU 0.02

#define BS 32

#define Ix(x, y) ((y) * NX + (x))

__global__ void init(float *p, float *u, float *v)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  p[Ix(i, j)] = 0;
  p[NX * NY + Ix(i, j)] = 0;
  u[Ix(i, j)] = 0;
  u[NX * NY + Ix(i, j)] = 0;
  v[Ix(i, j)] = 0;
  v[NX * NY + Ix(i, j)] = 0;
}

__global__ void step(float *p, float *u, float *v)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  float *pn = p + NY * NX;
  float *un = u + NY * NX;
  float *vn = v + NY * NX;
  
  if (i >= NX || j >= NY)
  {
    return;
  }

  float b = RHO * (1.0 / DT *
                       ((u[Ix(i + 1, j)] - u[Ix(i - 1, j)]) / (2 * DX) + (v[Ix(i, j + 1)] - v[Ix(i, j - 1)]) / (2 * DY)) -
                   powf((u[Ix(i + 1, j)] - u[Ix(i - 1, j)]) / (2 * DX), 2) -
                   2 * ((u[Ix(i, j + 1)] - u[Ix(i, j - 1)]) / (2 * DY) * (v[Ix(i + 1, j)] - v[Ix(i - 1, j)]) / (2 * DX)) -
                   powf((v[Ix(i, j + 1)] - v[Ix(i, j - 1)]) / (2 * DY), 2));

  // p
  for (int it = 0; it < NIT; it++)
  {
    pn[Ix(i, j)] = p[Ix(i, j)];

    __syncthreads();

    p[Ix(i, j)] = (powf(DY, 2) * (pn[Ix(i + 1, j)] + pn[Ix(i - 1, j)]) +
                   powf(DX, 2) * (pn[Ix(i, j + 1)] + pn[Ix(i, j - 1)]) -
                   b * powf(DX, 2) * powf(DY, 2)) /
                  (2 * (powf(DX, 2) + powf(DY, 2)));

    __syncthreads();
    if (j == 0)
    {
      p[Ix(i, 0)] = p[Ix(i, 1)];
    }
    else if (j == NY - 1)
    {
      p[Ix(i, NY - 1)] = p[Ix(i, NY - 2)];
    }
    else if (i == 0)
    {
      p[Ix(0, j)] = p[Ix(1, j)];
    }
    else if (i == NX - 1)
    {
      p[Ix(NX - 1, j)] = p[Ix(NX - 2, j)];
    }
  }

  // u,v
  un[Ix(i, j)] = u[Ix(i, j)];
  vn[Ix(i, j)] = v[Ix(i, j)];
  __syncthreads();
  if (j == 0)
  {
    u[Ix(i, 0)] = 0;
    v[Ix(i, 0)] = 0;
  }
  else if (j == NY - 1)
  {
    u[Ix(i, NY - 1)] = 1;
    v[Ix(i, NY - 1)] = 0;
  }
  else if (i == 0)
  {
    u[Ix(0, j)] = 0;
    v[Ix(0, j)] = 0;
  }
  else if (i == NX - 1)
  {
    u[Ix(NX - 1, j)] = 0;
    v[Ix(NX - 1, j)] = 0;
  }
  else
  {
    u[Ix(i, j)] = un[Ix(i, j)] - un[Ix(i, j)] * DT / DX * (un[Ix(i, j)] - un[Ix(i - 1, j)]) - vn[Ix(i, j)] * DT / DY * (un[Ix(i, j)] - un[Ix(i, j - 1)]) -
                  DT / (2 * RHO * DX) * (p[Ix(i + 1, j)] - p[Ix(i - 1, j)]) + NU * DT / powf(DX, 2) * (un[Ix(i + 1, j)] - 2 * un[Ix(i, j)] + un[Ix(i - 1, j)]) +
                  NU * DT / powf(DY, 2) * (un[Ix(i, j + 1)] - 2 * un[Ix(i, j)] + un[Ix(i, j - 1)]);

    v[Ix(i, j)] = vn[Ix(i, j)] - un[Ix(i, j)] * DT / DX * (vn[Ix(i, j)] - vn[Ix(i - 1, j)]) - vn[Ix(i, j)] * DT / DY * (vn[Ix(i, j)] - vn[Ix(i, j - 1)]) -
                  DT / (2 * RHO * DY) * (p[Ix(i, j + 1)] - p[Ix(i, j - 1)]) + NU * DT / powf(DX, 2) * (vn[Ix(i + 1, j)] - 2 * vn[Ix(i, j)] + vn[Ix(i - 1, j)]) +
                  NU * DT / powf(DY, 2) * (vn[Ix(i, j + 1)] - 2 * vn[Ix(i, j)] + vn[Ix(i, j - 1)]);
  }
}

int main()
{
  float *u, *v, *p;
  hipMallocManaged(&p, 2 * NX * NY * sizeof(float));
  hipMallocManaged(&u, 2 * NX * NY * sizeof(float));
  hipMallocManaged(&v, 2 * NX * NY * sizeof(float));
  init<<<dim3((NX + BS - 1) / BS, (NY + BS - 1) / BS, 1), dim3(BS, BS, 1)>>>(p, u, v);

  FILE *ufile = fopen("u.dat", "w");
  FILE *vfile = fopen("v.dat", "w");
  FILE *pfile = fopen("p.dat", "w");

  for (int n = 0; n < NT; n++)
  {

    step<<<dim3((NX + BS - 1) / BS, (NY + BS - 1) / BS, 1), dim3(BS, BS, 1)>>>(p, u, v);
    hipDeviceSynchronize();

    for (int j = 0; j < NY; j++)
    {
      for (int i = 0; i < NX; i++)
      {
        fprintf(ufile, "%f ", u[Ix(i, j)]);
      }
    }
    fprintf(ufile, "\n");
    for (int j = 0; j < NY; j++)
    {
      for (int i = 0; i < NX; i++)
      {
        fprintf(vfile, "%f ", v[Ix(i, j)]);
      }
    }
    fprintf(vfile, "\n");
    for (int j = 0; j < NY; j++)
    {
      for (int i = 0; i < NX; i++)
      {
        fprintf(pfile, "%f ", p[Ix(i, j)]);
      }
    }
    fprintf(pfile, "\n");
  }
  fclose(ufile);
  fclose(vfile);
  fclose(pfile);

  return 0;
}
